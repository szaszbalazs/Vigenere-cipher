#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <map>
#include <iostream>

#define TEXT_N 50000
#define KEY_M 15000
#define BLOCK_SIZE 256

__device__ char dev_TEXT[TEXT_N];
__device__ char dev_KEY[KEY_M];
__device__ char dev_ENCRYPTED[TEXT_N];
__device__ char dev_DECRYPTED[TEXT_N];

__shared__ char shr_TEXT[TEXT_N];
__shared__ char shr_KEY[KEY_M];



__device__ char vigenere_encrypt(char plaintext, char key, int key_length) {
	int p = plaintext - 'A'; // A=0, B=1, ..., Z=25
	int k = key - 'A'; // A=0, B=1, ..., Z=25
	int c = (p + k) % 26; // Titkosítás
	return c + 'A'; // Visszaalakítás karakterré
}

__device__ char vigenere_decrypt(char plaintext, char key, int key_length) {
	int p = plaintext - 'A'; // A=0, B=1, ..., Z=25
	int k = key - 'A'; // A=0, B=1, ..., Z=25
	int c = (p - k) % 26; // Titkosítás
	return c + 'A'; // Visszaalakítás karakterré
}

__global__ void Kernel()
{
	//shared -> ABC
	//A MÁR uppercased mondat is mehet sharedbe

	//maga a megoldás betűnként indul, minden szál megnézi, hogy az adott helyen [i] mit mivel kell összeadnia majd moduló
	//mituán ez megvan minden szál beírja magát a megfelelő helyre ->ehhez valszeg nem elég az empty string 
	//ha megvan vissza CPURA és kiírom

	//note to self: az uppercase átalakírást és a Kulcs fill-t végezhetné a GPU.
	//idő mérés, valamint occupancy calculator
	//atomi összeadás és kivonás amikor az encryptet csinálom illetve a decryptet.

	//3 féle
	//L1 cache és equals beállítása
	//konstans memória

	//első megoldás:
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (idx < TEXT_N) {
		char key_char = dev_KEY[idx % KEY_M];
		dev_ENCRYPTED[idx] = vigenere_encrypt(dev_TEXT[idx], key_char, KEY_M);
	}
	__syncthreads();

	if (idx < TEXT_N) {
		char key_char = dev_KEY[idx % KEY_M];
		dev_DECRYPTED[idx] = vigenere_decrypt(dev_ENCRYPTED[idx], key_char, KEY_M);
	}
}
__global__ void KernelOptimal() //Amennyire lehet shared memóriát használok olvasható adattagoknál
{
	

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < KEY_M)
	{
		shr_KEY[idx] = dev_KEY[idx];
	}

	if (idx < TEXT_N) {

		shr_TEXT[idx] = dev_TEXT[idx];

		char key_char = shr_KEY[idx % KEY_M];
		dev_ENCRYPTED[idx] = vigenere_encrypt(shr_TEXT[idx], key_char, KEY_M);
	}
	__syncthreads();

	if (idx < TEXT_N) {
		char key_char = shr_KEY[idx % KEY_M];
		dev_DECRYPTED[idx] = vigenere_decrypt(dev_ENCRYPTED[idx], key_char, KEY_M);
	}
}


int main()
{
	//N hosszúságú szöveg létrehozása
	//titkosítás
	//Az eredeti és a titkos szöveg alapján fel kell törni a kulcsot
	//atomi művelet és L1 cache, dinamikusan foglaljuk a memmóriát
	//sharedve mehet a constant adat
	//időmérés CPU, GPU

	std::string abc = "ABCDEFGHIJKLMNOPQRSTUVWXYZ ";

	//TODO: Random text [500] karakter és [10] kulcs
	std::string TEXT = "";
	std::string KEY = "";
	std::string KEY_C = "";
	std::string ENCRYPTED = "";
	std::string DECRYPTED = "";

	char TEXTARRAY[TEXT_N];
	char KEY_ARRAY[KEY_M];
	char ENCRYPTED_ARRAY[TEXT_N];
	char DECRYPTED_ARRAY[TEXT_N];

	float t;

	for (int i = 0; i < TEXT_N; i++)
	{
		char ch = 'a' + rand() % 26;
		TEXT += ch;
	}

	for (int i = 0; i < KEY_M; i++)
	{
		char ch = 'a' + rand() % 26;
		KEY += ch;
	}

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventRecord(start, 0);

	std::map<char, int> CharToInt;
	std::map<int, char> IntToChar;

	for (int i = 0; i <= abc.length(); i++)
	{
		CharToInt.insert({ (char)abc[i],i });
		IntToChar.insert({ i,(char)abc[i] });
	}

	//Kulcs fillelés és nagybetűssé alakítása
	for (int i = 0; i < TEXT.length(); i++)
	{

		for (int j = 0; j < KEY.length(); j++)
		{
			if (KEY_C.length()<TEXT.length())
			{
				KEY_C += (char)toupper(KEY[j]);

			}
		}
	}

	//Kulcs nagybetűssé alakítása
	for (int i = 0; i < KEY_M; i++)
	{
		KEY_ARRAY[i] = (char)toupper(KEY[i]);
	}

	//CPU MEGVALÓSÍTÁS
	//TEXT nagybetűssé alakítása
	for (int i = 0; i < TEXT.length(); i++)
	{
		TEXT[i] = (char)toupper(TEXT[i]);
		TEXTARRAY[i] = TEXT[i];
	}
	
	//encryption
	for (int i = 0; i < TEXT.length(); i++)
	{

		int charValue = CharToInt.at(TEXT[i]) + CharToInt.at(KEY_C[i]);
		int maradekos = charValue % (abc.length());

		//std::cout << CharToInt.at(TEXT[i]) << " ";
		ENCRYPTED += IntToChar.at(maradekos);
	}
	/*
	std::cout << " " << std::endl;
	for (int i = 0; i < TEXT.length(); i++)
	{

		int charValue = CharToInt.at(TEXT[i]) + CharToInt.at(KEY_C[i]);
		int maradekos = charValue % (abc.length());

		std::cout << CharToInt.at(KEY_C[i]) << " ";
		//ENCRYPTED += IntToChar.at(maradekos);
	}
	std::cout << " " << std::endl;
	for (int i = 0; i < TEXT.length(); i++)
	{

		int charValue = CharToInt.at(TEXT[i]) + CharToInt.at(KEY_C[i]);
		int maradekos = charValue % (abc.length());

		std::cout << charValue << " ";
		//ENCRYPTED += IntToChar.at(maradekos);
	}
	*/


	//decrypt
	for (int i = 0; i < ENCRYPTED.length(); i++)
	{
		int charValue = CharToInt.at(ENCRYPTED[i]) - CharToInt.at(KEY_C[i]);
		int maradekos = charValue % (abc.length());

		DECRYPTED += IntToChar.at(maradekos);
	}

	hipEventCreate(&stop);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&t, start, stop);
	
	//outputCPU
	/*std::cout << "Original Text: ";
	for (int i = 0; i < TEXT.length(); i++)
	{
		std::cout << TEXT[i];
	}
	std::cout << " "<<std::endl;
	std::cout << " " << std::endl;
	std::cout << "Encrypted Text: ";
	for (int i = 0; i < ENCRYPTED.length(); i++)
	{
		std::cout << ENCRYPTED[i];
	}
	std::cout << " " << std::endl;
	std::cout << " " << std::endl;
	std::cout << "Decrypted Text: ";
	for (int i = 0; i < DECRYPTED.length(); i++)
	{
		std::cout << DECRYPTED[i];
	}
	std::cout << " " << std::endl;*/
	std::cout << " " << std::endl;
	std::cout << "Time elapsed: ";
	std::cout << t << " ms";
	std::cout << " " << std::endl;

	//GPU
	//---------------------------
	hipMemcpyToSymbol(HIP_SYMBOL(dev_KEY),KEY_ARRAY,KEY_M * sizeof(char));
	hipMemcpyToSymbol(HIP_SYMBOL(dev_TEXT),TEXTARRAY,TEXT_N * sizeof(char));
	
	hipEvent_t start1, stop1;
	hipEventCreate(&start1);
	hipEventRecord(start1, 0);
	float t1;

	int block_count = (TEXT_N - 1) / BLOCK_SIZE + 1;

	Kernel << <block_count,BLOCK_SIZE >> > ();

	hipEventCreate(&stop1);
	hipEventRecord(stop1, 0);
	hipEventSynchronize(stop1);
	hipEventElapsedTime(&t1, start1, stop1);

	//itt még lehet gatya
	hipMemcpyFromSymbol(ENCRYPTED_ARRAY,HIP_SYMBOL(dev_ENCRYPTED),TEXT_N * sizeof(char));
	hipMemcpyFromSymbol(DECRYPTED_ARRAY, HIP_SYMBOL(dev_DECRYPTED), TEXT_N * sizeof(char));

	std::string EredmenyGPU1 = "";
	std::string EredmenyGPU2 = "";

	std::string EredmenyGPU3 = "";
	std::string EredmenyGPU4 = "";

	for (int i = 0; i < TEXT_N; i++)
	{
		EredmenyGPU1 += ENCRYPTED_ARRAY[i];
		EredmenyGPU2 += DECRYPTED_ARRAY[i];
	}



	std::cout << "-------------------------------------------------------------------------";
	std::cout << std::endl;
	std::cout << "GPUS megoldas #1"<<std::endl;
	std::cout << std::endl;
	/*std::cout << "Encrypted Text: " << EredmenyGPU1 <<std::endl;
	std::cout << std::endl;
	std::cout << "Decrypted Text: " << EredmenyGPU2 << std::endl;
	std::cout << std::endl;*/
	std::cout << "Time elapsed: " << t1 <<" ms" << std::endl;
	std::cout << std::endl;


	//Megoldások a második részre:

	char OptimalEncrypted[TEXT_N];
	char OptimalDecrypted[TEXT_N];

	//dst,src,count, cuda
	
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(KernelOptimal), hipFuncCachePreferEqual);
	float t3;

	hipMemcpyToSymbol(HIP_SYMBOL(dev_KEY), KEY_ARRAY, KEY_M * sizeof(char));
	hipMemcpyToSymbol(HIP_SYMBOL(dev_TEXT), TEXTARRAY, TEXT_N * sizeof(char));

	hipEvent_t start2, stop2;
	hipEventCreate(&start2);
	hipEventRecord(start2, 0);
	
	//Harmadik paraméter a extern shared memóriához kell.
	KernelOptimal << <block_count, BLOCK_SIZE>> > ();

	hipEventCreate(&stop2);
	hipEventRecord(stop2, 0);
	hipEventSynchronize(stop2);
	hipEventElapsedTime(&t3, start2, stop2);

	hipMemcpyFromSymbol(OptimalEncrypted, HIP_SYMBOL(dev_ENCRYPTED), TEXT_N * sizeof(char));
	hipMemcpyFromSymbol(OptimalDecrypted, HIP_SYMBOL(dev_DECRYPTED), TEXT_N * sizeof(char));


	for (int i = 0; i < TEXT_N; i++)
	{
		EredmenyGPU3 += OptimalEncrypted[i];
		EredmenyGPU4 += OptimalDecrypted[i];
	}
	;
	//GPU optimalizált megoldása
	std::cout << "-------------------------------------------------------------------------";
	std::cout << std::endl;
	std::cout << "GPUS megoldas #2 (Optimal)" << std::endl;
	std::cout << std::endl;
	/*std::cout << "Encrypted Text: " << EredmenyGPU3 << std::endl;
	std::cout << std::endl;
	std::cout << "Decrypted Text: " << EredmenyGPU4 << std::endl;*/
	//std::cout << std::endl;
	std::cout << "Time elapsed: " << t3 << " ms" << std::endl;
	std::cout << std::endl;
	std::cout << std::endl;

	//maxthreadsinblock:1024,49152shm,65536regspblock,65536 total const memory
	hipDeviceProp_t a;
	hipGetDeviceProperties(&a,0);
	;
}

